#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "gdeflate.h"
#include "nvcomp/gdeflate.h"

#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string.h>
#include <string>
#include <thrust/device_vector.h>
#include <vector>

#ifdef ENABLE_GDEFLATE
#define CHECK_NVCOMP_STATUS(status)                                            \
  if ((status) != nvcompSuccess)                                               \
    throw std::runtime_error("Failed to decompress data");
#else
#define CHECK_NVCOMP_STATUS(status)                                            \
  if ((status) != nvcompSuccess)                                               \
    throw std::runtime_error("nvcomp not configured with gdeflate support");
#endif

#define CUDA_CHECK(func)                                                       \
  do {                                                                         \
    hipError_t rt = (func);                                                   \
    if (rt != hipSuccess) {                                                   \
      std::cout << "API call failure \"" #func "\" with " << rt << " at "      \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      throw;                                                                   \
    }                                                                          \
  } while (0);

static size_t compute_batch_size(
    const std::vector<std::vector<char>>& data, const size_t chunk_size)
{
  size_t batch_size = 0;
  for (size_t i = 0; i < data.size(); ++i) {
    const size_t num_chunks = (data[i].size() + chunk_size - 1) / chunk_size;
    batch_size += num_chunks;
  }

  return batch_size;
}

std::vector<size_t> compute_chunk_sizes(
    const std::vector<std::vector<char>>& data,
    const size_t batch_size,
    const size_t chunk_size)
{
  std::vector<size_t> sizes(batch_size, chunk_size);

  size_t offset = 0;
  for (size_t i = 0; i < data.size(); ++i) {
    const size_t num_chunks = (data[i].size() + chunk_size - 1) / chunk_size;
    offset += num_chunks;
    if (data[i].size() % chunk_size != 0) {
      sizes[offset-1] = data[i].size() % chunk_size;
    }
  }
  return sizes;
}

std::vector<void*> get_input_ptrs(
    const std::vector<std::vector<char>>& data,
    const size_t batch_size,
    const size_t chunk_size)
{
  std::vector<void*> input_ptrs(batch_size);
  size_t chunk = 0;
  for (size_t i = 0; i < data.size(); ++i) {
    const size_t num_chunks = (data[i].size() + chunk_size - 1) / chunk_size;
    for (size_t j = 0; j < num_chunks; ++j)
      input_ptrs[chunk++] = const_cast<void*>(static_cast<const void*>(data[i].data() + j*chunk_size));
  }
  return input_ptrs;
}

class BatchDataCPU
{
public:
  BatchDataCPU(
      const std::vector<std::vector<char>>& host_data,
      const size_t chunk_size) :
      m_ptrs(),
      m_sizes(),
      m_data(),
      m_size(0)
  {
    m_size = compute_batch_size(host_data, chunk_size);
    m_sizes = compute_chunk_sizes(host_data, m_size, chunk_size);

    size_t data_size = std::accumulate(m_sizes.begin(), m_sizes.end(), static_cast<size_t>(0));
    m_data = std::vector<uint8_t>(data_size);

    size_t offset = 0;
    m_ptrs = std::vector<void*>(size());
    for (size_t i = 0; i < size(); ++i) {
      m_ptrs[i] = data() + offset;
      offset += m_sizes[i];
    }

    std::vector<void*> src = get_input_ptrs(host_data, size(), chunk_size);
    for (size_t i = 0; i < size(); ++i)
      std::memcpy(m_ptrs[i], src[i], m_sizes[i]);
  }

  BatchDataCPU(const size_t max_output_size, const size_t batch_size) :
      m_ptrs(),
      m_sizes(),
      m_data(),
      m_size(batch_size)
  {
    m_data = std::vector<uint8_t>(max_output_size * size());

    m_sizes = std::vector<size_t>(size(), max_output_size);

    m_ptrs = std::vector<void*>(batch_size);
    for (size_t i = 0; i < batch_size; ++i) {
      m_ptrs[i] = data() + max_output_size * i;
    }
  }

  BatchDataCPU(BatchDataCPU&& other) = default;

  // disable copying
  BatchDataCPU(const BatchDataCPU& other) = delete;
  BatchDataCPU& operator=(const BatchDataCPU& other) = delete;

  uint8_t* data() { return m_data.data(); }
  const uint8_t* data() const { return m_data.data(); }

  void** ptrs() { return m_ptrs.data(); }
  const void* const * ptrs() const { return m_ptrs.data(); }

  size_t* sizes() { return m_sizes.data(); }
  const size_t* sizes() const { return m_sizes.data(); }

  size_t size() const { return m_size; }

private:
  std::vector<void*> m_ptrs;
  std::vector<size_t> m_sizes;
  std::vector<uint8_t> m_data;
  size_t m_size;
};

class BatchData
{
public:
  BatchData(
      const std::vector<std::vector<char>>& host_data,
      const size_t chunk_size) :
      m_ptrs(),
      m_sizes(),
      m_data(),
      m_size(0)
  {
    m_size = compute_batch_size(host_data, chunk_size);

    m_data = thrust::device_vector<uint8_t>(chunk_size * size());

    std::vector<void*> uncompressed_ptrs(size());
    for (size_t i = 0; i < size(); ++i) {
      uncompressed_ptrs[i] = static_cast<void*>(data() + chunk_size * i);
    }

    m_ptrs = thrust::device_vector<void*>(uncompressed_ptrs);
    std::vector<size_t> sizes
        = compute_chunk_sizes(host_data, size(), chunk_size);
    m_sizes = thrust::device_vector<size_t>(sizes);

    // copy data to GPU
    size_t offset = 0;
    for (size_t i = 0; i < host_data.size(); ++i) {
      CUDA_CHECK(hipMemcpy(
          uncompressed_ptrs[offset],
          host_data[i].data(),
          host_data[i].size(),
          hipMemcpyHostToDevice));

      const size_t num_chunks
          = (host_data[i].size() + chunk_size - 1) / chunk_size;
      offset += num_chunks;
    }
  }

  BatchData(const BatchDataCPU& batch_data, bool copy_data=false) :
    m_ptrs(),
    m_sizes(),
    m_data(),
    m_size()
  {
    m_size = batch_data.size();
    m_sizes = thrust::device_vector<size_t>(batch_data.sizes(), batch_data.sizes() + size());

    size_t data_size = std::accumulate(batch_data.sizes(), batch_data.sizes() + size(), static_cast<size_t>(0));
    m_data = thrust::device_vector<uint8_t>(data_size);

    size_t offset = 0;
    std::vector<void*> ptrs(size());
    for (size_t i = 0; i < size(); ++i) {
      ptrs[i] = data() + offset;
      offset += batch_data.sizes()[i];
    }
    m_ptrs = thrust::device_vector<void*>(ptrs);

    if (copy_data) {
      const void* const * src = batch_data.ptrs();
      const size_t* bytes = batch_data.sizes();
      for (size_t i = 0; i < size(); ++i)
        CUDA_CHECK(hipMemcpy(ptrs[i], src[i], bytes[i], hipMemcpyHostToDevice));
    }
  }

  BatchData(const size_t max_output_size, const size_t batch_size) :
      m_ptrs(),
      m_sizes(),
      m_data(),
      m_size(batch_size)
  {
    m_data = thrust::device_vector<uint8_t>(max_output_size * size());

    std::vector<size_t> sizes(size(), max_output_size);
    m_sizes = thrust::device_vector<size_t>(sizes);

    std::vector<void*> ptrs(batch_size);
    for (size_t i = 0; i < batch_size; ++i) {
      ptrs[i] = data() + max_output_size * i;
    }
    m_ptrs = thrust::device_vector<void*>(ptrs);
  }

  BatchData(BatchData&& other) = default;

  // disable copying
  BatchData(const BatchData& other) = delete;
  BatchData& operator=(const BatchData& other) = delete;

  uint8_t* data() { return m_data.data().get(); }
  const uint8_t* data() const { return m_data.data().get(); }

  void** ptrs() { return m_ptrs.data().get(); }
  const void* const * ptrs() const { return m_ptrs.data().get(); }

  size_t* sizes() { return m_sizes.data().get(); }
  const size_t* sizes() const { return m_sizes.data().get(); }

  size_t size() const { return m_size; }

private:
  thrust::device_vector<void*> m_ptrs;
  thrust::device_vector<size_t> m_sizes;
  thrust::device_vector<uint8_t> m_data;
  size_t m_size;
};

inline bool operator==(
    const BatchDataCPU& lhs,
    const BatchData& rhs)
{
  size_t batch_size = lhs.size();

  if (lhs.size() != rhs.size()) return false;

  std::vector<size_t> rhs_sizes(rhs.size());
  CUDA_CHECK(hipMemcpy(rhs_sizes.data(), rhs.sizes(), rhs.size()*sizeof(size_t), hipMemcpyDeviceToHost));

  std::vector<void*> rhs_ptrs(rhs.size());
  CUDA_CHECK(hipMemcpy(rhs_ptrs.data(), rhs.ptrs(), rhs.size()*sizeof(void*), hipMemcpyDeviceToHost));

  for (size_t i = 0; i < batch_size; ++i) {
    if (lhs.sizes()[i] != rhs_sizes[i])
      return false;

    const uint8_t* lhs_ptr = reinterpret_cast<const uint8_t*>(lhs.ptrs()[i]);
    const uint8_t* rhs_ptr = reinterpret_cast<const uint8_t*>(rhs_ptrs[i]);
    std::vector<uint8_t> rhs_data(rhs_sizes[i]);
    CUDA_CHECK(hipMemcpy(rhs_data.data(), rhs_ptr, rhs_sizes[i], hipMemcpyDeviceToHost));

    for (size_t j = 0; j < rhs_sizes[i]; ++j)
      if (lhs_ptr[j] != rhs_data[j])
        return false;
  }

  return true;
}

// Benchmark performance from the binary data file fname
static void
run_example(const std::vector<std::vector<char>>& data)
{
  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;

  // build up input batch on CPU
  BatchDataCPU input_data_cpu(data, chunk_size);
  std::cout << "chunks: " << input_data_cpu.size() << std::endl;

  // compression

  // Get max output size per chunk
  nvcompError_t status;
  size_t max_out_bytes;
  status = nvcompBatchedGdeflateCompressGetMaxOutputChunkSize(
      chunk_size, &max_out_bytes);
  CHECK_NVCOMP_STATUS(status);

  // Allocate and prepare output/compressed batch
  BatchDataCPU compress_data_cpu(max_out_bytes, input_data_cpu.size());

#ifdef ENABLE_GDEFLATE
  // Compress on the CPU using gdeflate CPU batched API
  gdeflate::compressCPU(input_data_cpu.ptrs(),
      input_data_cpu.sizes(),
      chunk_size,
      input_data_cpu.size(),
      compress_data_cpu.ptrs(),
      compress_data_cpu.sizes());
#else
  throw std::runtime_error("nvcomp configured without gdeflate support. "
      "Please check the documentation for details on configuring nvcomp with gdeflate.")
#endif

  // compute compression ratio
  size_t* compressed_sizes_host = compress_data_cpu.sizes();
  size_t comp_bytes = 0;
  for (size_t i = 0; i < compress_data_cpu.size(); ++i)
    comp_bytes += compressed_sizes_host[i];

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;

  // Copy compressed data to GPU
  BatchData compress_data(compress_data_cpu, true);

  // Allocate and build up decompression batch on GPU
  BatchData decomp_data(input_data_cpu, false);

  // Create CUDA stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // CUDA events to measure decompression time
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  // gdeflate GPU decompression
  size_t decomp_temp_bytes;
  status = nvcompBatchedGdeflateDecompressGetTempSize(
      compress_data.size(), chunk_size, &decomp_temp_bytes);

  void* d_decomp_temp;
  CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));

  CUDA_CHECK(hipStreamSynchronize(stream));

  // Run decompression
  status = nvcompBatchedGdeflateDecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      chunk_size,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      stream);

  // Validate decompressed data against input
  if (!(input_data_cpu == decomp_data))
    throw std::runtime_error("Failed to validate decompressed data");
  else
    std::cout << "decompression validated :)" << std::endl;

  // Re-run decompression to get throughput
  hipEventRecord(start, stream);
  status = nvcompBatchedGdeflateDecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      chunk_size,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      stream);
  hipEventRecord(end, stream);

  CUDA_CHECK(hipStreamSynchronize(stream));
  CHECK_NVCOMP_STATUS(status);

  float ms;
  hipEventElapsedTime(&ms, start, end);

  double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
  std::cout << "decompression throughput (GB/s): "
            << decompression_throughput << std::endl;

  hipFree(d_decomp_temp);

  hipEventDestroy(start);
  hipEventDestroy(end);
  hipStreamDestroy(stream);
}
#undef CHECK_NVCOMP_STATUS

std::vector<char> readFile(const std::string& filename)
{
  std::vector<char> buffer(4096);
  std::vector<char> host_data;

  std::ifstream fin(filename, std::ifstream::binary);
  fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);

  size_t num;
  do {
    num = fin.readsome(buffer.data(), buffer.size());
    host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
  } while (num > 0);

  return host_data;
}

std::vector<std::vector<char>>
multi_file(const std::vector<std::string>& filenames)
{
  std::vector<std::vector<char>> split_data;

  for (auto const& filename : filenames) {
    split_data.emplace_back(readFile(filename));
  }

  return split_data;
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names(argc - 1);

  if (argc == 1) {
    std::cerr << "Must specify at least one file." << std::endl;
    return 1;
  }

  // if `-f` is speficieid, assume single file mode
  if (strcmp(argv[1], "-f") == 0) {
    if (argc == 2) {
      std::cerr << "Missing file name following '-f'" << std::endl;
      return 1;
    } else if (argc > 3) {
      std::cerr << "Unknown extra arguments with '-f'." << std::endl;
      return 1;
    }

    file_names = {argv[2]};
  } else {
    // multi-file mode
    for (int i = 1; i < argc; ++i) {
      file_names[i - 1] = argv[i];
    }
  }

  auto data = multi_file(file_names);

  run_example(data);

  return 0;
}
