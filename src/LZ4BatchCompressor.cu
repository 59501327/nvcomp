#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "CudaUtils.h"
#include "LZ4BatchCompressor.h"
#include "LZ4CompressionKernels.h"
#include "TempSpaceBroker.h"
#include "common.h"

namespace nvcomp
{

/******************************************************************************
 * KERNELS ********************************************************************
 *****************************************************************************/

namespace
{
template <int BLOCK_SIZE>
__global__ void collectItemSizes(
    size_t* const item_sizes,
    const size_t* const* const item_prefixes,
    const size_t* const uncomp_sizes,
    const size_t chunk_size,
    const size_t batch_size)
{
  const size_t item = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  if (item < batch_size) {
    const size_t num_chunks = roundUpDiv(uncomp_sizes[item], chunk_size);

    item_sizes[item] = item_prefixes[item][num_chunks];
  }
}

} // namespace

/******************************************************************************
 * HELPER FUNCTIONS ***********************************************************
 *****************************************************************************/

namespace
{

size_t compute_staging_bytes(
    const size_t* const decomp_data_size,
    const size_t batch_size,
    const size_t chunk_size)
{
  const size_t num_chunks
      = lz4ComputeChunksInBatch(decomp_data_size, batch_size, chunk_size);

  const size_t staging_bytes = roundUpTo(
      lz4CompressComputeTempSize(num_chunks, chunk_size), sizeof(size_t));

  return staging_bytes;
}

} // namespace

/******************************************************************************
 * PUBLIC STATIC METHODS ******************************************************
 *****************************************************************************/

size_t LZ4BatchCompressor::calculate_workspace_size(
    const size_t* const decomp_data_size,
    const size_t batch_size,
    const size_t chunk_size)
{
  const size_t staging_bytes
      = compute_staging_bytes(decomp_data_size, batch_size, chunk_size);
  const size_t pointer_bytes = 2 * batch_size * sizeof(uint8_t*);
  const size_t size_bytes
      = (batch_size * sizeof(size_t*)) + (batch_size * sizeof(size_t));
  const size_t offset_bytes = (batch_size + 1) * sizeof(size_t);

  return staging_bytes + pointer_bytes + size_bytes + offset_bytes;
}

/******************************************************************************
 * CONSTRUCTORS / DESTRUCTOR **************************************************
 *****************************************************************************/

LZ4BatchCompressor::LZ4BatchCompressor(
    const uint8_t* const* decomp_data,
    const size_t* decomp_data_size,
    const size_t batch_size,
    const size_t chunk_size) :
    m_batch_size(batch_size),
    m_chunk_size(chunk_size),
    m_pinned_input_sizes(decomp_data_size, decomp_data_size + batch_size),
    m_pinned_input_ptrs(decomp_data, decomp_data + batch_size),
    m_pinned_output_sizes(batch_size),
    m_pinned_output_ptrs(batch_size),
    m_pinned_output_offsets(batch_size),
    m_workspace(nullptr),
    m_workspace_size(0),
    m_host_item_sizes(nullptr),
    m_output_configured(false)
{
  // do nothing
}

/******************************************************************************
 * PUBLIC METHODS *************************************************************
 *****************************************************************************/

size_t LZ4BatchCompressor::get_workspace_size() const
{
  return calculate_workspace_size(
      m_pinned_input_sizes.data(), m_batch_size, m_chunk_size);
}

void LZ4BatchCompressor::configure_workspace(
    void* const workspace, const size_t size)
{
  const size_t required_Size = get_workspace_size();
  if (size < required_Size) {
    throw std::runtime_error(
        "Insufficient workspace size: " + std::to_string(size) + " / "
        + std::to_string(required_Size));
  } else if (workspace == nullptr) {
    throw std::runtime_error("Workspace cannot be null.");
  }

  m_workspace = workspace;
  m_workspace_size = size;
}

void LZ4BatchCompressor::configure_output(
    uint8_t* const* const device_locations,
    size_t* const* const device_sizes,
    const size_t* const device_offsets,
    size_t* const host_item_sizes)
{
  std::copy(
      device_sizes, device_sizes + m_batch_size, m_pinned_output_sizes.begin());
  std::copy(
      device_locations,
      device_locations + m_batch_size,
      m_pinned_output_ptrs.begin());
  std::copy(
      device_offsets,
      device_offsets + m_batch_size,
      m_pinned_output_offsets.begin());
  m_host_item_sizes = host_item_sizes;
  m_output_configured = true;
}

void LZ4BatchCompressor::compress_async(hipStream_t stream)
{
  if (!is_workspace_configured()) {
    throw std::runtime_error(
        "Workspace must be configured before compressing.");
  } else if (!is_output_configured()) {
    throw std::runtime_error("Output must be configured before compressing.");
  }

  TempSpaceBroker temp(m_workspace, m_workspace_size);

  uint8_t* workspace;
  const size_t workspace_size = compute_staging_bytes(
      m_pinned_input_sizes.data(), m_batch_size, m_chunk_size);
  temp.reserve(&workspace, workspace_size);

  // TODO: do this all in one copy
  const uint8_t** in_ptrs_device;
  temp.reserve(&in_ptrs_device, m_batch_size);
  CudaUtils::copy_async(
      in_ptrs_device, m_pinned_input_ptrs.data(), m_batch_size,
      HOST_TO_DEVICE,
      stream);

  size_t* in_sizes_device;
  temp.reserve(&in_sizes_device, m_batch_size);
  CudaUtils::copy_async(
      in_sizes_device, m_pinned_input_sizes.data(), m_batch_size,
      HOST_TO_DEVICE, stream);

  uint8_t** out_ptrs_device;
  temp.reserve(&out_ptrs_device, m_batch_size);
  CudaUtils::copy_async(
      out_ptrs_device, m_pinned_output_ptrs.data(), m_batch_size,
      HOST_TO_DEVICE, stream);

  size_t** out_sizes_device;
  temp.reserve(&out_sizes_device, m_batch_size);
  CudaUtils::copy_async(
      out_sizes_device, m_pinned_output_sizes.data(), m_batch_size,
      HOST_TO_DEVICE, stream);

  size_t* out_prefix_offsets_device;
  temp.reserve(&out_prefix_offsets_device, m_batch_size);
  CudaUtils::copy_async(
      out_prefix_offsets_device,
      m_pinned_output_offsets.data(),
      m_batch_size,
      HOST_TO_DEVICE,
      stream);

  // TODO: implement step_size
  lz4CompressBatch(
      in_ptrs_device,
      in_sizes_device,
      m_pinned_input_sizes.data(),
      m_batch_size,
      m_chunk_size,
      workspace,
      workspace_size,
      out_ptrs_device,
      out_sizes_device,
      out_prefix_offsets_device,
      stream);

  // repurpose in item sizes for collecting batch totals
  if (m_host_item_sizes) {
    size_t* item_sizes_device = in_sizes_device;

    // We're using 64 threads here to maximize the number of active thread
    // blocks we can have for a given number of items, as we would expect to
    // struggle to make use of much of the GPU with a number of items in the
    // hundreds or thousands.
    constexpr const int BLOCK_SIZE = 64;

    const dim3 grid(roundUpDiv(m_batch_size, BLOCK_SIZE));
    const dim3 block(BLOCK_SIZE);

    collectItemSizes<BLOCK_SIZE><<<grid, block, 0, stream>>>(
        item_sizes_device,
        out_sizes_device,
        in_sizes_device,
        m_chunk_size,
        m_batch_size);

    CudaUtils::copy_async(
        m_host_item_sizes, item_sizes_device, m_batch_size,
        DEVICE_TO_HOST, stream);
  }
}

/******************************************************************************
 * PRIVATE METHODS ************************************************************
 *****************************************************************************/

bool LZ4BatchCompressor::is_workspace_configured() const
{
  return m_workspace != nullptr;
}

bool LZ4BatchCompressor::is_output_configured() const
{
  return m_output_configured;
}

} // namespace nvcomp
